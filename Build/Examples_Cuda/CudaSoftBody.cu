#include "hip/hip_runtime.h"
#include "CudaSoftBody.cuh"

#define SQRT2 1.41421356237

inline float3 vec_to_float(Vector3 vec) {
	float3 out;
	out.x = vec.x;
	out.y = vec.y;
	out.z = vec.z;
	return out;
}

inline Vector3 float_to_vec(float3 flo) {
	return Vector3(flo.x, flo.y, flo.z);
}

__global__ void CollideParticles(float baumgarte_factor, uint num_particles, Node* particles, CPUParticle * cpu_particles, float separation, int num_CPU_obj)
{
	uint index = blockIdx.x*blockDim.x + threadIdx.x;
	if (index >= num_particles)
		return;

	Node p = particles[index];

	if (p.invmass != 0.0f)
	{
		for (int i = 0; i < num_CPU_obj; ++i) {
			CPUParticle cpu_p = cpu_particles[i];
			//float3 newpos = make_float3(cpu_p.pos.x, cpu_p.pos.y, cpu_p.pos.z);

			//Do a quick sphere-sphere test
			float3 ab = cpu_p.pos - p.pos;
			float lengthSq = dot(ab, ab);

			const float diameterSq = ((separation + cpu_p.radius + 1.0f) * (separation + cpu_p.radius + 1.0f));
			if (lengthSq < diameterSq)
			{
				//We have a collision!
				float len = sqrtf(lengthSq);
				float3 abn = ab / len;

				//Direct normal collision (no friction/shear)
				float abnVel = dot(cpu_p.vel - cpu_p.vel, abn);
				float jn = -(abnVel * (1.f + 0.00f));

				//Extra energy to overcome overlap error
				float overlap = cpu_p.radius - len;
				float b = overlap * baumgarte_factor;


				jn += b;

				jn = max(jn, 0.0f);
				p.vel -= abn * (jn * 0.5f);
			}
		}
	}
	particles[index] = p;
	
}

__global__ void UpdateSpringSet(Node * nodes, SpringSet set, int w, int h, float h_rest, float d_rest, float dt) {
	
	uint index = (blockIdx.x*blockDim.x) + threadIdx.x;

	int i = 0;
	int j = 0;

	while (index >= w) {
		index -= w;
		i += 1;
	}
	j = index;

	
	int p1_idx = 0;
	int p2_idx = 0;
	float rest_length = 0.0f;

	switch (set) {
	// ...
	// o--o  o--o  o--o ...
	// o--o  o--o  o--o ...
	case HOR1:
		if (j % 2 == 0 && j != w - 1) {
			rest_length = h_rest;
			p1_idx = (i*w) + j;
			p2_idx = (i*w) + j + 1;
		}
		else { return; }
		break;
	// o  o--o  o--o  o ...
	// o  o--o  o--o  o ...
	// ...
	case HOR2:
		if (j % 2 != 0 && j != w - 1) {
			rest_length = h_rest;
			p1_idx = (i*w) + j;
			p2_idx = (i*w) + j + 1;
		}
		else { return; }
		break;
	// o  o  o  o  o  o
	// |  |  |  |  |  |
	// o  o  o  o  o  o
	// 
	// o  o  o  o  o  o
	// |  |  |  |  |  |
	// o  o  o  o  o  o
	case VERT1:
		if (i % 2 == 0 && i != h - 1) {
			rest_length = h_rest;
			p1_idx = (i*w) + j;
			p2_idx = ((i+1)*w) + j;
		}
		else { return; }
		break;
	// o  o  o  o  o  o
	// 
	// o  o  o  o  o  o
	// |  |  |  |  |  |
	// o  o  o  o  o  o
	//
	// o  o  o  o  o  o
	case VERT2:
		if (i % 2 != 0 && i != h - 1) {
			rest_length = h_rest;
			p1_idx = (i*w) + j;
			p2_idx = ((i+1)*w) + j;
		}
		else { return; }
		break;
	// o  o  o  o  o  o
	//  \  \  \  \  \
	// o  o  o  o  o  o
	// 
	// o  o  o  o  o  o
	//  \  \  \  \  \
	// o  o  o  o  o  o
	case DIAG_L1:
		if (i % 2 == 0 && i != h - 1 && j != 0) {
			rest_length = d_rest;
			p1_idx = (i*w) + j;
			p2_idx = ((i+1)*w) + j - 1;
		}
		else { return; }
		break;
	// o  o  o  o  o  o
	// 
	// o  o  o  o  o  o
	//  \  \  \  \  \ 
	// o  o  o  o  o  o
	//
	// o  o  o  o  o  o
	case DIAG_L2:
		if (i % 2 != 0 && i != h - 1 && j != 0) {
			rest_length = d_rest;
			p1_idx = (i*w) + j;
			p2_idx = ((i+1)*w) + j - 1;
		}
		else { return; }
		break;
	// o  o  o  o  o  o
	//  /  /  /  /  /
	// o  o  o  o  o  o
	// 
	// o  o  o  o  o  o
	//   /  /  /  /  /
	// o  o  o  o  o  o
	case DIAG_R1:
		if (i % 2 == 0 && i != h - 1 && j != w-1) {
			rest_length = d_rest;
			p1_idx = (i*w) + j;
			p2_idx = ((i+1)*w) + j + 1;
		}
		else { return; }
		break;
	// o  o  o  o  o  o
	//  
	// o  o  o  o  o  o
	//	/  /  /  /  /
	// o  o  o  o  o  o
	//
	// o  o  o  o  o  o
	case DIAG_R2:
		if (i % 2 != 0 && i != h - 1 && j != w - 1) {
			rest_length = d_rest;
			p1_idx = (i*w) + j;
			p2_idx = ((i+1)*w) + j + 1;
		}
		else { return; }
		break;
	}

	if (p1_idx != p2_idx) {
		float3 ab = nodes[p2_idx].pos - nodes[p1_idx].pos;
		float3 abn = normalize(ab);

		float abnVel = dot(nodes[p1_idx].vel - nodes[p2_idx].vel, abn);

		float constraintMass = nodes[p1_idx].invmass + nodes[p2_idx].invmass;

		if (constraintMass > 0.0f) {
			float b = 0.0f;

			float distance_offset = length(ab) - rest_length;
			float baumgarte_scalar = 0.1f;
			b = -(baumgarte_scalar
				/ dt)
				* distance_offset;

			float jn = (distance_offset * 0.6f) / (constraintMass * dt) - (0.01f * (abnVel));
			//s.p1.vel -= abn * jn * s.p1.invmass;
			//s.p2.vel += abn * jn * s.p2.invmass;

			nodes[p1_idx].vel += abn * jn * nodes[p1_idx].invmass;
			nodes[p2_idx].vel -= abn * jn * nodes[p2_idx].invmass;
		}
	}


}

__host__
struct UpdatePositions
{
	UpdatePositions(float dt, float3 gravity)
		: _dt(dt)
		, _gravity(gravity)
	{
	}

	float _dt;
	float3 _gravity;

	__host__ __device__
		void operator()(Node& p)
	{
		//Time integration
		if(p.invmass != 0)
			p.vel += _gravity;
		p.vel *= 0.999f;

		p.pos += p.vel * _dt;
	}
};

CudaSoftBody::CudaSoftBody(int w, int h, float s, Vector3 pos, GLuint tex)
{
	this->w = w;
	this->h = h;
	this->s = s;
	this->pos = new Vector3(pos);

	nodes = new Node[w*h];

	for (int i = 0; i < h; ++i) {
		for (int j = 0; j < w; ++j) {
			Node pnode;
			pnode.pos = vec_to_float((Vector3(pos.x + (s*j), pos.y, pos.z + (s*i))));
			pnode.vel = make_float3(0.0f, 0.0f, 0.0f);
			pnode.invmass = (60.0f);
			if (i == 0)
				pnode.invmass = 0.f;
			
			nodes[(i*w) + j] = pnode;

			
		}
	}

	straight_rest_length = s;
	diagonal_rest_length = SQRT2*s;

	RenderNode* rnode = new RenderNode();

	m = Mesh::GenerateMesh(w - 1, h - 1, s);
	m->SetTexture(tex);
	RenderNode* dummy = new RenderNode(m, Vector4(1, 0, 0, 1));
	rnode->AddChild(dummy);
	dummy->SetCulling(false);

	rnode->SetTransform(Matrix4::Translation(Vector3(pos.x, pos.y, pos.z)));
	rnode->SetBoundingRadius(10);

	obj = rnode;

	gpuErrchk(hipMalloc(&cuda_nodes,		w * h *	sizeof(Node)));
	
}

CudaSoftBody::~CudaSoftBody()
{
}

void CudaSoftBody::UpdateMesh() {
	int k = 0;
	for (int i = 0; i < h - 1; ++i) {
		for (int j = 0; j < w - 1; ++j) {
			m->vertices[k] = float_to_vec(nodes[(i * w) + j].pos) - *pos;
			m->vertices[k + 1] = float_to_vec(nodes[(i * w) + j + 1].pos) - *pos;
			m->vertices[k + 2] = float_to_vec(nodes[((i + 1) * w) + j].pos) - *pos;

			m->vertices[k + 3] = float_to_vec(nodes[(i * w) + j + 1].pos) - *pos;
			m->vertices[k + 4] = float_to_vec(nodes[((i + 1) * w) + j + 1].pos) - *pos;
			m->vertices[k + 5] = float_to_vec(nodes[((i + 1) * w) + j].pos) - *pos;

			k += 6;
		}
	}
	m->GenerateNormals();
	m->GenerateTangents();
	m->ClearBuffers();
	m->BufferData();
}

void CudaSoftBody::UpdateSoftBody(float dt, vector<PhysicsNode *> cpuparticles)
{
	const float fixed_timestep = 1.0f / 60.0f;
	float baumgarte_factor = 0.05f / fixed_timestep;
	const float3 gravity = make_float3(0, -0.01f, 0);

	cuda_cpu_particles = new CPUParticle[cpuparticles.size()];
	cpu_particles = new CPUParticle[cpuparticles.size()];

	for (int i = 0; i < cpuparticles.size(); ++i) {
		cpu_particles[i].invmass	= cpuparticles[i]->GetInverseMass();
		cpu_particles[i].radius		= cpuparticles[i]->GetBoundingRadius();
		cpu_particles[i].pos		= vec_to_float(cpuparticles[i]->GetPosition());
		cpu_particles[i].vel		= vec_to_float(cpuparticles[i]->GetLinearVelocity());
	}


	gpuErrchk(hipMalloc(&cuda_cpu_particles, cpuparticles.size() * sizeof(CPUParticle)));

	gpuErrchk(hipMemcpy(cuda_cpu_particles, cpu_particles, cpuparticles.size() * sizeof(CPUParticle), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(cuda_nodes, nodes, w * h * sizeof(Node), hipMemcpyHostToDevice));

	thrust::for_each(
		thrust::device_ptr<Node>(cuda_nodes),
		thrust::device_ptr<Node>(cuda_nodes + (w*h)),
		UpdatePositions(fixed_timestep, gravity));



	

	for (int i = 0; i < 10; ++i) {
		CollideParticles << < w*h, 1 >> > (baumgarte_factor, w*h, cuda_nodes, cuda_cpu_particles, s, cpuparticles.size());

		UpdateSpringSet << < w*h, 1 >> > (cuda_nodes, HOR1, w, h, straight_rest_length, diagonal_rest_length, dt);

		UpdateSpringSet << < w*h, 1 >> > (cuda_nodes, HOR2, w, h, straight_rest_length, diagonal_rest_length, dt);

		UpdateSpringSet << < w*h, 1 >> > (cuda_nodes, VERT1, w, h, straight_rest_length, diagonal_rest_length, dt);

		UpdateSpringSet << < w*h, 1 >> > (cuda_nodes, VERT2, w, h, straight_rest_length, diagonal_rest_length, dt);

		UpdateSpringSet << < w, h >> > (cuda_nodes, DIAG_R1, w, h, straight_rest_length, diagonal_rest_length, dt);

		UpdateSpringSet << < w, h >> > (cuda_nodes, DIAG_R2, w, h, straight_rest_length, diagonal_rest_length, dt);

		UpdateSpringSet << < w, h >> > (cuda_nodes, DIAG_L1, w, h, straight_rest_length, diagonal_rest_length, dt);

		UpdateSpringSet << < w, h >> > (cuda_nodes, DIAG_L2, w, h, straight_rest_length, diagonal_rest_length, dt);

	}

	gpuErrchk(hipMemcpy(nodes, cuda_nodes, w * h * sizeof(Node), hipMemcpyDeviceToHost));
	gpuErrchk(hipFree(cuda_cpu_particles));
	cuda_cpu_particles = NULL;
	UpdateMesh();
}

