#include "hip/hip_runtime.h"
#include "CudaSoftBody.cuh"

#define SQRT2 1.41421356237

inline float3 vec_to_float(Vector3 vec) {
	float3 out;
	out.x = vec.x;
	out.y = vec.y;
	out.z = vec.z;
	return out;
}

inline Vector3 float_to_vec(float3 flo) {
	return Vector3(flo.x, flo.y, flo.z);
}


__device__ void resolveSpring(Node * nodes, Spring s,float dt) {

}


__global__ void UpdateSpringSet(Node * nodes, SpringSet set, int w, int h, float h_rest, float d_rest, float dt) {
	
	uint index = (blockIdx.x*blockDim.x) + threadIdx.x;

	int i = 0;
	int j = 0;

	while (index >= w) {
		index -= w;
		i += 1;
	}
	j = index;

	
	int p1_idx = 0;
	int p2_idx = 0;
	float rest_length = 0.0f;

	switch (set) {
	// ...
	// o--o  o--o  o--o ...
	// o--o  o--o  o--o ...
	case HOR1:
		if (j % 2 == 0 && j != w - 1) {
			rest_length = h_rest;
			p1_idx = (i*w) + j;
			p2_idx = (i*w) + j + 1;
		}
		else { return; }
		break;
	// o  o--o  o--o  o ...
	// o  o--o  o--o  o ...
	// ...
	case HOR2:
		if (j % 2 != 0 && j != w - 1) {
			rest_length = h_rest;
			p1_idx = (i*w) + j;
			p2_idx = (i*w) + j + 1;
		}
		else { return; }
		break;
	// o  o  o  o  o  o
	// |  |  |  |  |  |
	// o  o  o  o  o  o
	// 
	// o  o  o  o  o  o
	// |  |  |  |  |  |
	// o  o  o  o  o  o
	case VERT1:
		if (i % 2 == 0 && i != h - 1) {
			rest_length = h_rest;
			p1_idx = (i*w) + j;
			p2_idx = ((i+1)*w) + j;
		}
		else { return; }
		break;
	// o  o  o  o  o  o
	// 
	// o  o  o  o  o  o
	// |  |  |  |  |  |
	// o  o  o  o  o  o
	//
	// o  o  o  o  o  o
	case VERT2:
		if (i % 2 != 0 && i != h - 1) {
			rest_length = h_rest;
			p1_idx = (i*w) + j;
			p2_idx = ((i+1)*w) + j;
		}
		else { return; }
		break;
	// o  o  o  o  o  o
	//  \  \  \  \  \
	// o  o  o  o  o  o
	// 
	// o  o  o  o  o  o
	//  \  \  \  \  \
	// o  o  o  o  o  o
	case DIAG_L1:
		if (i % 2 == 0 && i != h - 1 && j != 0) {
			rest_length = d_rest;
			p1_idx = (i*w) + j;
			p2_idx = ((i+1)*w) + j - 1;
		}
		else { return; }
		break;
	// o  o  o  o  o  o
	// 
	// o  o  o  o  o  o
	//  \  \  \  \  \ 
	// o  o  o  o  o  o
	//
	// o  o  o  o  o  o
	case DIAG_L2:
		if (i % 2 != 0 && i != h - 1 && j != 0) {
			rest_length = d_rest;
			p1_idx = (i*w) + j;
			p2_idx = ((i+1)*w) + j - 1;
		}
		else { return; }
		break;
	// o  o  o  o  o  o
	//  /  /  /  /  /
	// o  o  o  o  o  o
	// 
	// o  o  o  o  o  o
	//   /  /  /  /  /
	// o  o  o  o  o  o
	case DIAG_R1:
		if (i % 2 == 0 && i != h - 1 && j != w-1) {
			rest_length = d_rest;
			p1_idx = (i*w) + j;
			p2_idx = ((i+1)*w) + j + 1;
		}
		else { return; }
		break;
	// o  o  o  o  o  o
	//  
	// o  o  o  o  o  o
	//	/  /  /  /  /
	// o  o  o  o  o  o
	//
	// o  o  o  o  o  o
	case DIAG_R2:
		if (i % 2 != 0 && i != h - 1 && j != w - 1) {
			rest_length = d_rest;
			p1_idx = (i*w) + j;
			p2_idx = ((i+1)*w) + j + 1;
		}
		else { return; }
		break;
	}

	if (p1_idx != p2_idx) {
		float3 ab = nodes[p2_idx].pos - nodes[p1_idx].pos;
		float3 abn = normalize(ab);

		float abnVel = dot(nodes[p1_idx].vel - nodes[p2_idx].vel, abn);

		float constraintMass = nodes[p1_idx].invmass + nodes[p2_idx].invmass;

		if (constraintMass > 0.0f) {
			float b = 0.0f;

			float distance_offset = length(ab) - rest_length;
			float baumgarte_scalar = 0.1f;
			b = -(baumgarte_scalar
				/ dt)
				* distance_offset;

			float jn = ((distance_offset * 0.1f) / (constraintMass * dt)) - (0.01f * (abnVel));
			//s.p1.vel -= abn * jn * s.p1.invmass;
			//s.p2.vel += abn * jn * s.p2.invmass;

			nodes[p1_idx].vel += abn * jn * nodes[p1_idx].invmass;
			nodes[p2_idx].vel -= abn * jn * nodes[p2_idx].invmass;
		}
	}


}




CudaSoftBody::CudaSoftBody(int w, int h, float s, Vector3 pos, GLuint tex)
{
	this->w = w;
	this->h = h;
	this->pos = new Vector3(pos);
	for (int i = 0; i < h; ++i) {
		for (int j = 0; j < w; ++j) {
			PhysicsNode * pnode = new PhysicsNode();
			pnode->SetPosition(Vector3(pos.x + (s*j), pos.y + (s*i), pos.z));

			pnode->SetInverseMass(60.0f);

			//if (i == h - 1) {//&& j == 0 || (i == h - 1 && j == w - 1)) {
			//	pnode->SetInverseMass(0.0f);
			//}

			CollisionShape* pColshape = new SphereCollisionShape(0.7f*s);
			pnode->SetCollisionShape(pColshape);
			physicsnodes.push_back(pnode);

		}
	}

	straight_rest_length = s;
	diagonal_rest_length = SQRT2*s;

	RenderNode* rnode = new RenderNode();

	m = Mesh::GenerateMesh(w - 1, h - 1, s);
	m->SetTexture(tex);
	RenderNode* dummy = new RenderNode(m, Vector4(1, 0, 0, 1));
	rnode->AddChild(dummy);
	dummy->SetCulling(false);

	rnode->SetTransform(Matrix4::Translation(Vector3(pos.x, pos.y, pos.z)));
	rnode->SetBoundingRadius(10);

	mgo = new MultiGameObject("softBody", rnode, physicsnodes, this->pos);
	mgo->SetSiblingsCollide(false);
	mgo->RotateObject(Vector3(1, 0, 0), -90);
	mgo->UpdatePosition(Vector3(2.5, 15, -2.5));

	ScreenPicker::Instance()->RegisterNodeForMouseCallback(
		dummy, //Dummy is the rendernode that actually contains the drawable mesh, and the one we can to 'drag'
		std::bind(&CommonUtils::DragableObjectCallback, mgo, std::placeholders::_1, std::placeholders::_2, std::placeholders::_3, std::placeholders::_4)
	);

	physicsnodes[0]->SetOnUpdateCallback(
		std::bind(
			&CudaSoftBody::UpdateMesh,
			this,
			std::placeholders::_1)
	);

	nodes = new Node[physicsnodes.size()];

	for (int i = 0; i < physicsnodes.size(); ++i) {
		nodes[i].invmass = physicsnodes[i]->GetInverseMass();
		nodes[i].pos = vec_to_float(physicsnodes[i]->GetPosition());
		nodes[i].vel = vec_to_float(physicsnodes[i]->GetLinearVelocity());
	}

	gpuErrchk(hipMalloc(&cuda_nodes,		w * h *	sizeof(Node)));
	
}

CudaSoftBody::~CudaSoftBody()
{
}

void CudaSoftBody::UpdateMesh(const Matrix4 &matrix) {
	int k = 0;
	for (int i = 0; i < h - 1; ++i) {
		for (int j = 0; j < w - 1; ++j) {
			m->vertices[k] = physicsnodes[(i * w) + j]->GetPosition() - *pos;
			m->vertices[k + 1] = physicsnodes[(i * w) + j + 1]->GetPosition() - *pos;
			m->vertices[k + 2] = physicsnodes[((i + 1) * w) + j]->GetPosition() - *pos;

			m->vertices[k + 3] = physicsnodes[(i * w) + j + 1]->GetPosition() - *pos;
			m->vertices[k + 4] = physicsnodes[((i + 1) * w) + j + 1]->GetPosition() - *pos;
			m->vertices[k + 5] = physicsnodes[((i + 1) * w) + j]->GetPosition() - *pos;

			k += 6;
		}
	}
	m->GenerateNormals();
	m->GenerateTangents();
	m->ClearBuffers();
	m->BufferData();
}

void CudaSoftBody::UpdateSoftBody(float dt)
{

	for (int i = 0; i < physicsnodes.size(); ++i) {
		nodes[i].invmass = physicsnodes[i]->GetInverseMass();
		nodes[i].pos = vec_to_float(physicsnodes[i]->GetPosition());
		nodes[i].vel = vec_to_float(physicsnodes[i]->GetLinearVelocity());
	}

	gpuErrchk(hipMemcpy(cuda_nodes, nodes, w * h * sizeof(Node), hipMemcpyHostToDevice));

	UpdateSpringSet << < w*h, 1 >> > (cuda_nodes, HOR1, w, h, straight_rest_length, diagonal_rest_length, dt);

	UpdateSpringSet << < w*h, 1 >> > (cuda_nodes, HOR2, w, h, straight_rest_length, diagonal_rest_length, dt);

	UpdateSpringSet << < w*h, 1 >> > (cuda_nodes, VERT1, w, h, straight_rest_length, diagonal_rest_length, dt);

	UpdateSpringSet << < w*h, 1 >> > (cuda_nodes, VERT2, w, h, straight_rest_length, diagonal_rest_length, dt);

	UpdateSpringSet << < w, h >> > (cuda_nodes, DIAG_R1, w, h,	straight_rest_length, diagonal_rest_length, dt);

	UpdateSpringSet << < w, h >> > (cuda_nodes, DIAG_R2, w, h,	straight_rest_length, diagonal_rest_length, dt);
	
	UpdateSpringSet << < w, h >> > (cuda_nodes, DIAG_L1, w, h,	straight_rest_length, diagonal_rest_length, dt);

	UpdateSpringSet << < w, h >> > (cuda_nodes, DIAG_L2, w, h,	straight_rest_length, diagonal_rest_length, dt);

	gpuErrchk(hipMemcpy(nodes, cuda_nodes, w * h * sizeof(Node), hipMemcpyDeviceToHost));

	for (int i = 0; i < physicsnodes.size(); ++i) {
		physicsnodes[i]->SetLinearVelocity(float_to_vec(nodes[i].vel));
	}
}


